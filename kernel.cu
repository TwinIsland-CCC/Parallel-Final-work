#include "hip/hip_runtime.h"
#pragma warning(disable : 4996)

#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<ctime>
#include<bitset>
#include<vector>
#include<string>
#include<sstream>

#include<unordered_set>

#include<bitset>

#include<windows.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include<immintrin.h>  // AVX

#include<mutex>
#include<condition_variable>

using namespace std;

#define THREAD_NUM 8  // 线程数量
const int BLOCK_SIZE = 1024;


//--------------------------------------函数声明--------------------------------------


int n = 0;  // 矩阵大小
const int k = 1;

int ek_num;  // 非零消元子个数
int et_num;  // 导入被消元行行数

/* 所有数据规模：
1:  130   22    8
2:  254   106   53
3:  562   170   53
4:  1011  539   263
5:  2362  1226  453
6:  3799  2759  1953
7:  8399  6375  4535
8:  23075 18748 14325
9:  39060 23904 14921
10: 43577 39477 54274
11: 85401 5724  756
*/

//------------------------------------数据导入工具------------------------------------
const int column_num_c = 2362;
const int ek_num_c = 1226;  // 非零消元子个数
const int et_num_c = 453;  // 导入被消元行行数

string dir = "C:/Users/CCC/source/repos/GrobnerGE/GrobnerGE/data/t5/";
stringstream ss;


int bit_size = column_num_c / 32 + 1;
class MyBitSet {
public:
	int head;  // 首项
	int* content;

	MyBitSet() {
		head = -1; content = new int[bit_size];
		for (int i = 0; i < bit_size; i++) content[i] = 0;
	}

	// bool operator[](size_t index) {}

	MyBitSet& operator^=(const MyBitSet& b) {  // 默认两个输入bitset长度相同
		for (int i = 0; i < bit_size; i++) content[i] ^= b.content[i];
		for (int i = 0; i < bit_size; i++)
		{
			for (int j = 0; j < 32; j++)
			{
				if ((content[i] & (1 << j)))
				{
					head = i * 32 + j;
					return *this;
				}
			}
		}
		head = -1;
		return *this;
	}

	MyBitSet& my_xor_AVX(const MyBitSet& b) {
		__m256i v_this, v_b;
		int i = 0;
		for (i; i < bit_size - 8; i += 8) {
			v_this = _mm256_loadu_si256((__m256i*) & content[i]);
			v_b = _mm256_loadu_si256((__m256i*) & b.content[i]);
			v_this = _mm256_xor_si256(v_this, v_b);
			_mm256_storeu_si256((__m256i*) & content[i], v_this);
		}
		for (i; i < bit_size; i++)
		{
			content[i] ^= b.content[i];
		}
		for (int i = 0; i < bit_size; i++)
		{
			for (int j = 0; j < 32; j++)
			{
				if ((content[i] & (1 << j)))
				{
					head = i * 32 + j;
					return *this;
				}
			}
		}
		head = -1;
		return *this;
	}


	int test(int index) {
		return content[index / 32] & (1 << (index % 32)) ? 1 : 0;  // 寻址方式
	}

	void set(int index) {  // 置位
		content[index / 32] |= (1 << (index % 32));
	}

	bool any() {
		for (int i = 0; i < bit_size; i++) if (content[i]) return true;
		return false;
	}

private:

};


bitset<column_num_c> eks_c[column_num_c];  // 消元子，开大一些便于检索与升格
bitset<column_num_c> ets_c[et_num_c];

int lp_eks_c[column_num_c];
int lp_ets_c[et_num_c];

int** eks_bits = new int* [column_num_c];
int** ets_bits = new int* [et_num_c];




MyBitSet eks[column_num_c];
MyBitSet ets[et_num_c];



long long head, tail, freq;


//------------------------------------输出调试函数------------------------------------

void reverse_output_MyB()
{
	ofstream outp(dir + "output_MyB.txt");
	for (int i = 0; i < et_num_c; i++)
	{
		for (int j = 0; j < n; j++) if (ets[i].test(j)) outp << n - j - 1 << " ";
		outp << endl;
	}
	outp.close();
}

//------------------------------------数据读取函数------------------------------------

void readData_reverse_MyB()
{  // 倒序读入数据，读入静态位集
	string inek, inet;
	stringstream ss_inek, ss_inet;
	ifstream inElimKey(dir + "elimkey.txt");  // 消元子
	ifstream inElimTar(dir + "elimtar.txt");  // 被消元行
	int inek_loc, p_ek = 0, inet_loc, p_et = 0;  // 用于数据读入
	int lp = -1;
	while (true)  // 读取消元子
	{
		getline(inElimKey, inek);
		ss_inek = stringstream(inek);
		while (ss_inek >> inek)
		{
			inek_loc = stoi(inek);
			if (lp == -1)
			{
				lp = column_num_c - inek_loc - 1;
				eks[lp].head = lp;
			}
			//cout << inek_loc << " ";
			eks[lp].set(column_num_c - inek_loc - 1);

		}
		lp = -1;  p_ek++;
		if (inek.empty()) break;
		//cout << eks_c[p_ek] << endl;
	}
	//cout << "ek_complete" << endl;

	while (true)  // 读取被消元行
	{
		getline(inElimTar, inet);
		ss_inet = stringstream(inet);
		while (ss_inet >> inet)
		{
			inet_loc = stoi(inet);
			if (lp == -1)
			{
				lp = column_num_c - inet_loc - 1;
				ets[p_et].head = lp;

			}
			//cout << inet_loc << " ";
			ets[p_et].set(column_num_c - inet_loc - 1);
		}
		lp = -1;  p_et++;
		if (inet.empty()) break;
		//cout << ets_c[p_et] << endl;
	}
	//cout << "et_complete" << endl;
	inElimKey.close();
	inElimTar.close();
	//cout << "init_complete" << endl;

}

void init_MyB() {
	readData_reverse_MyB();  // 逆序初始化消元子和被消元行阵列
	cout << "init_complete" << endl;
}



//-----------------------------------CUDA算法核函数-----------------------------------

__global__ void upshift_kernel(int i, int** gpu_eks, int** gpu_ets, int* gpu_lp_ets, int et_num_c) {
	//主线程做
	if (blockIdx.x == 0 && threadIdx.x == 0)
	for (int j = 0; j < et_num_c; j++)
	{
		if (i == gpu_lp_ets[j])  // 说明存在对应被消元行
		{
			gpu_eks[i] = gpu_ets[j];
			gpu_lp_ets[j] = -1;
			break;
		}
	}
}


__global__ void elim_kernel(int i, int** gpu_ek, int** gpu_et, int* gpu_lp_ets, int et_num_c, int column_num_c, int bit_size) {
	//int tx = blockDim.x * blockIdx.x + threadIdx.x;
	//if (!tx) data[k * N + k] = 1.0;//对角线元素设为 1
	//int row = k + 1 + blockIdx.x;//每个块负责一行
	//float t;
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockIdx.x;//每个块负责一个被消元行

	bool find = false;
	for (int j = row; j < et_num_c; j += blockDim.x)  // 循环划分并行化
	{
		if (i == gpu_lp_ets[j])  // 说明存在对应被消元行
		{
			//gpu_et[j] ^= gpu_ek[i];

			for (int k = 0; k < bit_size; k++) gpu_et[j][k] ^= gpu_ek[i][k];
			__syncthreads();
			if (threadIdx.x == 0)
			{
				for (int k = 0; k < bit_size; k++)
				{
					for (int l = 0; l < 32; l++)
					{
						if ((gpu_et[j][k] & (1 << l)))
						{
							gpu_lp_ets[j] = k * 32 + l;
							find = true;
							break;
						}
					}
					if (find)
					{
						find ^= find;
						break;
					}
					else gpu_lp_ets[j] = -1;
				}
			}
		}
	}
}



//-------------------------------------展示所有内容-------------------------------------



//------------------------------------CUDA消去算法------------------------------------

void CUDA_GE(bitset<column_num_c>* ek, bitset<column_num_c>* et, int* lp_ets) {

	for (int i = 0; i < column_num_c; i++)
	{	
		eks_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
		{
			eks_bits[i][j] = 0;
		}
	}
	for (int i = 0; i < et_num_c; i++)
	{
		ets_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
		{
			ets_bits[i][j] = 0;
		}
	}
	string inek, inet;
	stringstream ss_inek, ss_inet;
	ifstream inElimKey(dir + "elimkey.txt");  // 消元子
	ifstream inElimTar(dir + "elimtar.txt");  // 被消元行
	int inek_loc, p_ek = 0, inet_loc, p_et = 0;  // 用于数据读入
	int lp = -1;
	while (true)  // 读取消元子
	{
		getline(inElimKey, inek);
		ss_inek = stringstream(inek);
		while (ss_inek >> inek)
		{
			inek_loc = stoi(inek);
			if (lp == -1)
			{
				lp = column_num_c - inek_loc - 1;
				//eks[lp].head = lp;
				lp_eks_c[lp] = lp;
			}
			//cout << inek_loc << " ";
			//eks[lp].set(column_num_c - inek_loc - 1);
			// 拆解set函数
			eks_bits[lp][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;  p_ek++;
		if (inek.empty()) break;
		//cout << eks_c[p_ek] << endl;
	}
	//cout << "ek_complete" << endl;

	while (true)  // 读取被消元行
	{
		getline(inElimTar, inet);
		ss_inet = stringstream(inet);
		while (ss_inet >> inet)
		{
			inet_loc = stoi(inet);
			if (lp == -1)
			{
				lp = column_num_c - inet_loc - 1;
				//ets[p_et].head = lp;
				lp_ets_c[p_et] = lp;
			}
			//cout << inet_loc << " ";
			//ets[p_et].set(column_num_c - inet_loc - 1);
			ets_bits[p_et][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;  p_et++;
		if (inet.empty()) break;
		//cout << ets_c[p_et] << endl;
	}
	//cout << "et_complete" << endl;
	inElimKey.close();
	inElimTar.close();
	//cout << "init_complete" << endl;

	//show_1d(temp, N);
	hipError_t ret;//用于错误检查，当 CUDA 接口调用成功会返回 cudaSucess



	int** gpu_ets;
	int** gpu_eks;

	int** gpu_ets_bits = new int* [et_num_c];  // 存放的是指针首地址
	int** gpu_eks_bits = new int* [column_num_c];  // 存放的是指针首地址
	int* gpu_lp_ets;




	int gpu_et_size = et_num_c * sizeof(int*);
	int gpu_ek_size = column_num_c * sizeof(int*);

	int bitset_size = sizeof(int) * bit_size;
	int gpu_lp_ets_size = et_num_c * sizeof(int);

	for (int i = 0; i < et_num; i++)
	{
		int* bits;  // 暂时存放显存地址
		if (hipMalloc(&bits, gpu_et_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, ets_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");
		gpu_ets_bits[i] = bits;
	}

	if (hipMalloc(&gpu_ets, gpu_et_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_ets, gpu_ets_bits, gpu_et_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	for (int i = 0; i < column_num_c; i++)
	{
		int* bits;  // 暂时存放显存地址
		if (hipMalloc(&bits, gpu_ek_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, eks_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");
		gpu_eks_bits[i] = bits;
	}

	if (hipMalloc(&gpu_eks, gpu_ek_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_eks, gpu_eks_bits, gpu_ek_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	//分配显存空间并且进行错误检查
	if (hipMalloc(&gpu_lp_ets, gpu_lp_ets_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	
	//将数据传输至 GPU 端并进行错误检查
	if (hipMemcpy(gpu_lp_ets, lp_ets_c, gpu_lp_ets_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	dim3 dimBlock(BLOCK_SIZE, 1), dimGrid(1, 1); //线程块、线程网格

	hipEvent_t start, stop;  //计时器
	float elapsedTime = 0.0;
	hipEventCreate(&start), hipEventCreate(&stop);
	hipEventRecord(start, 0);  //开始计时

	hipError_t exec;
	for (int i = 0; i < column_num_c; i++)  // 取每个消元子，对被消元行进行操作，便于并行化
	{
		if (!(eks_bits[i][i / 32] & (1 << (i % 32))))  // 消元子被逆序初始化时满足“行号” = “首项”的条件
		{
			upshift_kernel << <1, 1 >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c);
		}
		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		exec = hipGetLastError();
		if (exec != hipSuccess) printf("upshift_kernel failed, %s\n", hipGetErrorString(exec));

		elim_kernel << <dimGrid, dimBlock >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c, column_num_c, bit_size);//负责消去任务的核函数

		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		exec = hipGetLastError();
		if (exec != hipSuccess) printf("elim_kernel failed, %s\n", hipGetErrorString(exec));
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//停止计时
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("CUDA_GE:%f ms\n", elapsedTime);

	hipError_t cudaStatus2 = hipGetLastError();
	if (hipGetLastError() != hipSuccess) fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
	//将数据传回 CPU 端并进行错误检查
	//if (hipMemcpy(ek, gpu_ek, gpu_ek_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");
	//if (hipMemcpy(et, gpu_et, gpu_et_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");
	//if (hipMemcpy(lp_ets, gpu_lp_ets, gpu_et_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");

	//reverse_output_c();

	//show_1d(result, N);  // 测试
	//show_in_file_1d(result, N);  // 测试

	//hipFree(gpu_eks);//释放显存空间，用 CUDA 接口分配的空间必须用 hipFree 释放

	//销毁计时器
	hipEventDestroy(start);
	hipEventDestroy(stop);
}


int main() {
	//freopen("input.txt", "r", stdin);
	//float* temp = new float[N * N];
	//for (int i = 0; i < N; i++) for (int j = 0; j < N; j++) cin >> temp[i * N + j];

	cout << "矩阵大小为" << column_num_c << "，消元子个数为" << ek_num_c << "，被消元行行数为" << et_num_c << endl;


	//-----------------------------------------------------------------
	//float* t_1d = generate_1d(N);
	CUDA_GE(eks_c, ets_c, lp_ets_c);

	//-----------------------------------------------------------------

	//system("pause");
}
#pragma warning(disable : 4996)

#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<ctime>
#include<bitset>
#include<vector>
#include<string>
#include<sstream>

#include<unordered_set>

#include<bitset>

#include<windows.h>

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include<immintrin.h>  // AVX

#include<mutex>
#include<condition_variable>

using namespace std;

#define THREAD_NUM 8  // 线程数量
const int BLOCK_SIZE = 1024;


//--------------------------------------函数声明--------------------------------------


int n = 0;  // 矩阵大小
const int k = 1;

int ek_num;  // 非零消元子个数
int et_num;  // 导入被消元行行数

/* 所有数据规模：
1:  130   22    8
2:  254   106   53
3:  562   170   53
4:  1011  539   263
5:  2362  1226  453
6:  3799  2759  1953
7:  8399  6375  4535
8:  23075 18748 14325
9:  39060 23904 14921
10: 43577 39477 54274
11: 85401 5724  756
*/

//------------------------------------数据导入工具------------------------------------
const int column_num_c = 2362;
const int ek_num_c = 1226;  // 非零消元子个数
const int et_num_c = 453;  // 导入被消元行行数

string dir = "C:/Users/CCC/source/repos/GrobnerGE/GrobnerGE/data/t5/";
stringstream ss;


int bit_size = column_num_c / 32 + 1;
class MyBitSet {
public:
	int head;  // 首项
	int* content;

	MyBitSet() {
		head = -1; content = new int[bit_size];
		for (int i = 0; i < bit_size; i++) content[i] = 0;
	}

	// bool operator[](size_t index) {}

	MyBitSet& operator^=(const MyBitSet& b) {  // 默认两个输入bitset长度相同
		for (int i = 0; i < bit_size; i++) content[i] ^= b.content[i];
		for (int i = 0; i < bit_size; i++)
		{
			for (int j = 0; j < 32; j++)
			{
				if ((content[i] & (1 << j)))
				{
					head = i * 32 + j;
					return *this;
				}
			}
		}
		head = -1;
		return *this;
	}

	MyBitSet& my_xor_AVX(const MyBitSet& b) {
		__m256i v_this, v_b;
		int i = 0;
		for (i; i < bit_size - 8; i += 8) {
			v_this = _mm256_loadu_si256((__m256i*) & content[i]);
			v_b = _mm256_loadu_si256((__m256i*) & b.content[i]);
			v_this = _mm256_xor_si256(v_this, v_b);
			_mm256_storeu_si256((__m256i*) & content[i], v_this);
		}
		for (i; i < bit_size; i++)
		{
			content[i] ^= b.content[i];
		}
		for (int i = 0; i < bit_size; i++)
		{
			for (int j = 0; j < 32; j++)
			{
				if ((content[i] & (1 << j)))
				{
					head = i * 32 + j;
					return *this;
				}
			}
		}
		head = -1;
		return *this;
	}


	int test(int index) {
		return content[index / 32] & (1 << (index % 32)) ? 1 : 0;  // 寻址方式
	}

	void set(int index) {  // 置位
		content[index / 32] |= (1 << (index % 32));
	}

	bool any() {
		for (int i = 0; i < bit_size; i++) if (content[i]) return true;
		return false;
	}

private:

};


bitset<column_num_c> eks_c[column_num_c];  // 消元子，开大一些便于检索与升格
bitset<column_num_c> ets_c[et_num_c];

int lp_eks_c[column_num_c];
int lp_ets_c[et_num_c];

int** eks_bits = new int* [column_num_c];
int** ets_bits = new int* [et_num_c];




MyBitSet eks[column_num_c];
MyBitSet ets[et_num_c];



long long head, tail, freq;


//------------------------------------输出调试函数------------------------------------

void reverse_output_MyB()
{
	ofstream outp(dir + "output_MyB.txt");
	for (int i = 0; i < et_num_c; i++)
	{
		for (int j = 0; j < n; j++) if (ets[i].test(j)) outp << n - j - 1 << " ";
		outp << endl;
	}
	outp.close();
}

//------------------------------------数据读取函数------------------------------------

void readData_reverse_MyB()
{  // 倒序读入数据，读入静态位集
	string inek, inet;
	stringstream ss_inek, ss_inet;
	ifstream inElimKey(dir + "elimkey.txt");  // 消元子
	ifstream inElimTar(dir + "elimtar.txt");  // 被消元行
	int inek_loc, p_ek = 0, inet_loc, p_et = 0;  // 用于数据读入
	int lp = -1;
	while (true)  // 读取消元子
	{
		getline(inElimKey, inek);
		ss_inek = stringstream(inek);
		while (ss_inek >> inek)
		{
			inek_loc = stoi(inek);
			if (lp == -1)
			{
				lp = column_num_c - inek_loc - 1;
				eks[lp].head = lp;
			}
			//cout << inek_loc << " ";
			eks[lp].set(column_num_c - inek_loc - 1);

		}
		lp = -1;  p_ek++;
		if (inek.empty()) break;
		//cout << eks_c[p_ek] << endl;
	}
	//cout << "ek_complete" << endl;

	while (true)  // 读取被消元行
	{
		getline(inElimTar, inet);
		ss_inet = stringstream(inet);
		while (ss_inet >> inet)
		{
			inet_loc = stoi(inet);
			if (lp == -1)
			{
				lp = column_num_c - inet_loc - 1;
				ets[p_et].head = lp;

			}
			//cout << inet_loc << " ";
			ets[p_et].set(column_num_c - inet_loc - 1);
		}
		lp = -1;  p_et++;
		if (inet.empty()) break;
		//cout << ets_c[p_et] << endl;
	}
	//cout << "et_complete" << endl;
	inElimKey.close();
	inElimTar.close();
	//cout << "init_complete" << endl;

}

void init_MyB() {
	readData_reverse_MyB();  // 逆序初始化消元子和被消元行阵列
	cout << "init_complete" << endl;
}



//-----------------------------------CUDA算法核函数-----------------------------------

__global__ void upshift_kernel(int i, int** gpu_eks, int** gpu_ets, int* gpu_lp_ets, int et_num_c) {
	//主线程做
	if (blockIdx.x == 0 && threadIdx.x == 0)
	for (int j = 0; j < et_num_c; j++)
	{
		if (i == gpu_lp_ets[j])  // 说明存在对应被消元行
		{
			gpu_eks[i] = gpu_ets[j];
			gpu_lp_ets[j] = -1;
			break;
		}
	}
}


__global__ void elim_kernel(int i, int** gpu_ek, int** gpu_et, int* gpu_lp_ets, int et_num_c, int column_num_c, int bit_size) {
	//int tx = blockDim.x * blockIdx.x + threadIdx.x;
	//if (!tx) data[k * N + k] = 1.0;//对角线元素设为 1
	//int row = k + 1 + blockIdx.x;//每个块负责一行
	//float t;
	int tx = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockIdx.x;//每个块负责一个被消元行

	bool find = false;
	for (int j = row; j < et_num_c; j += blockDim.x)  // 循环划分并行化
	{
		if (i == gpu_lp_ets[j])  // 说明存在对应被消元行
		{
			//gpu_et[j] ^= gpu_ek[i];

			for (int k = 0; k < bit_size; k++) gpu_et[j][k] ^= gpu_ek[i][k];
			__syncthreads();
			if (threadIdx.x == 0)
			{
				for (int k = 0; k < bit_size; k++)
				{
					for (int l = 0; l < 32; l++)
					{
						if ((gpu_et[j][k] & (1 << l)))
						{
							gpu_lp_ets[j] = k * 32 + l;
							find = true;
							break;
						}
					}
					if (find)
					{
						find ^= find;
						break;
					}
					else gpu_lp_ets[j] = -1;
				}
			}
		}
	}
}



//-------------------------------------展示所有内容-------------------------------------



//------------------------------------CUDA消去算法------------------------------------

void CUDA_GE(bitset<column_num_c>* ek, bitset<column_num_c>* et, int* lp_ets) {

	for (int i = 0; i < column_num_c; i++)
	{	
		eks_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
		{
			eks_bits[i][j] = 0;
		}
	}
	for (int i = 0; i < et_num_c; i++)
	{
		ets_bits[i] = new int[bit_size];
		for (int j = 0; j < bit_size; j++)
		{
			ets_bits[i][j] = 0;
		}
	}
	string inek, inet;
	stringstream ss_inek, ss_inet;
	ifstream inElimKey(dir + "elimkey.txt");  // 消元子
	ifstream inElimTar(dir + "elimtar.txt");  // 被消元行
	int inek_loc, p_ek = 0, inet_loc, p_et = 0;  // 用于数据读入
	int lp = -1;
	while (true)  // 读取消元子
	{
		getline(inElimKey, inek);
		ss_inek = stringstream(inek);
		while (ss_inek >> inek)
		{
			inek_loc = stoi(inek);
			if (lp == -1)
			{
				lp = column_num_c - inek_loc - 1;
				//eks[lp].head = lp;
				lp_eks_c[lp] = lp;
			}
			//cout << inek_loc << " ";
			//eks[lp].set(column_num_c - inek_loc - 1);
			// 拆解set函数
			eks_bits[lp][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;  p_ek++;
		if (inek.empty()) break;
		//cout << eks_c[p_ek] << endl;
	}
	//cout << "ek_complete" << endl;

	while (true)  // 读取被消元行
	{
		getline(inElimTar, inet);
		ss_inet = stringstream(inet);
		while (ss_inet >> inet)
		{
			inet_loc = stoi(inet);
			if (lp == -1)
			{
				lp = column_num_c - inet_loc - 1;
				//ets[p_et].head = lp;
				lp_ets_c[p_et] = lp;
			}
			//cout << inet_loc << " ";
			//ets[p_et].set(column_num_c - inet_loc - 1);
			ets_bits[p_et][(column_num_c - inek_loc - 1) / 32] |= (1 << ((column_num_c - inek_loc - 1) % 32));
		}
		lp = -1;  p_et++;
		if (inet.empty()) break;
		//cout << ets_c[p_et] << endl;
	}
	//cout << "et_complete" << endl;
	inElimKey.close();
	inElimTar.close();
	//cout << "init_complete" << endl;

	//show_1d(temp, N);
	hipError_t ret;//用于错误检查，当 CUDA 接口调用成功会返回 cudaSucess



	int** gpu_ets;
	int** gpu_eks;

	int** gpu_ets_bits = new int* [et_num_c];  // 存放的是指针首地址
	int** gpu_eks_bits = new int* [column_num_c];  // 存放的是指针首地址
	int* gpu_lp_ets;




	int gpu_et_size = et_num_c * sizeof(int*);
	int gpu_ek_size = column_num_c * sizeof(int*);

	int bitset_size = sizeof(int) * bit_size;
	int gpu_lp_ets_size = et_num_c * sizeof(int);

	for (int i = 0; i < et_num; i++)
	{
		int* bits;  // 暂时存放显存地址
		if (hipMalloc(&bits, gpu_et_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, ets_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");
		gpu_ets_bits[i] = bits;
	}

	if (hipMalloc(&gpu_ets, gpu_et_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_ets, gpu_ets_bits, gpu_et_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	for (int i = 0; i < column_num_c; i++)
	{
		int* bits;  // 暂时存放显存地址
		if (hipMalloc(&bits, gpu_ek_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
		if (hipMemcpy(bits, eks_bits[i], bitset_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");
		gpu_eks_bits[i] = bits;
	}

	if (hipMalloc(&gpu_eks, gpu_ek_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	if (hipMemcpy(gpu_eks, gpu_eks_bits, gpu_ek_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	//分配显存空间并且进行错误检查
	if (hipMalloc(&gpu_lp_ets, gpu_lp_ets_size) != hipSuccess)  printf("hipMalloc gpudata failed!\n");
	
	//将数据传输至 GPU 端并进行错误检查
	if (hipMemcpy(gpu_lp_ets, lp_ets_c, gpu_lp_ets_size, hipMemcpyHostToDevice) != hipSuccess) printf("hipMemcpyHostToDevice failed!\n");

	dim3 dimBlock(BLOCK_SIZE, 1), dimGrid(1, 1); //线程块、线程网格

	hipEvent_t start, stop;  //计时器
	float elapsedTime = 0.0;
	hipEventCreate(&start), hipEventCreate(&stop);
	hipEventRecord(start, 0);  //开始计时

	hipError_t exec;
	for (int i = 0; i < column_num_c; i++)  // 取每个消元子，对被消元行进行操作，便于并行化
	{
		if (!(eks_bits[i][i / 32] & (1 << (i % 32))))  // 消元子被逆序初始化时满足“行号” = “首项”的条件
		{
			upshift_kernel << <1, 1 >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c);
		}
		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		exec = hipGetLastError();
		if (exec != hipSuccess) printf("upshift_kernel failed, %s\n", hipGetErrorString(exec));

		elim_kernel << <dimGrid, dimBlock >> > (i, gpu_eks, gpu_ets, gpu_lp_ets, et_num_c, column_num_c, bit_size);//负责消去任务的核函数

		hipDeviceSynchronize();//CPU 与 GPU 之间的同步函数
		exec = hipGetLastError();
		if (exec != hipSuccess) printf("elim_kernel failed, %s\n", hipGetErrorString(exec));
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);//停止计时
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("CUDA_GE:%f ms\n", elapsedTime);

	hipError_t cudaStatus2 = hipGetLastError();
	if (hipGetLastError() != hipSuccess) fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
	//将数据传回 CPU 端并进行错误检查
	//if (hipMemcpy(ek, gpu_ek, gpu_ek_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");
	//if (hipMemcpy(et, gpu_et, gpu_et_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");
	//if (hipMemcpy(lp_ets, gpu_lp_ets, gpu_et_size, hipMemcpyDeviceToHost) != hipSuccess) printf("hipMemcpyDeviceToHost failed!\n");

	//reverse_output_c();

	//show_1d(result, N);  // 测试
	//show_in_file_1d(result, N);  // 测试

	//hipFree(gpu_eks);//释放显存空间，用 CUDA 接口分配的空间必须用 hipFree 释放

	//销毁计时器
	hipEventDestroy(start);
	hipEventDestroy(stop);
}


int main() {
	//freopen("input.txt", "r", stdin);
	//float* temp = new float[N * N];
	//for (int i = 0; i < N; i++) for (int j = 0; j < N; j++) cin >> temp[i * N + j];

	cout << "矩阵大小为" << column_num_c << "，消元子个数为" << ek_num_c << "，被消元行行数为" << et_num_c << endl;


	//-----------------------------------------------------------------
	//float* t_1d = generate_1d(N);
	CUDA_GE(eks_c, ets_c, lp_ets_c);

	//-----------------------------------------------------------------

	//system("pause");
}
